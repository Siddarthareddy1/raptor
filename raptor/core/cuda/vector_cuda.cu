#include "hip/hip_runtime.h"
// Copyright (c) 2015-2017, RAPtor Developer Team
// License: Simplified BSD, http://opensource.org/licenses/BSD-2-Clause
#include "vector_cuda.hpp"
#include "vector_cuda_kernels.cuh"

using namespace raptor;

/**************************************************************
*****   Vector Set Constant Value
**************************************************************
***** Initializes the vector to a constant value
*****
***** Parameters
***** -------------
***** alpha : data_t
*****    Constant value to set each element of vector to
**************************************************************/
void Vector::set_const_value(data_t alpha)
{
    printf("before cuda kernel\n");
    set_const_value_kernel<<<tblocks,blocksize>>>(alpha, dev_ptr, values.size());
    printf("after cuda kernel\n");
}

/**************************************************************
*****   Vector Set Random Values
**************************************************************
***** Initializes each element of the vector to a random
***** value
**************************************************************/
void Vector::set_rand_values(int seed)
{
    set_rand_values_kernel<<<tblocks,blocksize>>>(seed, dev_ptr, values.size());
}

/**************************************************************
*****   Vector Copy
**************************************************************
***** Copies each vector value of y into values
*****
***** Parameters
***** -------------
***** y : Vector&
*****    Vector to be copied.  Must have same local rows
*****    and same first row
**************************************************************/
void Vector::copy(const Vector& y)
{
    num_values = y.num_values;
    b_vecs = y.b_vecs;
    values.resize(num_values * b_vecs);
    std::copy(y.values.begin(), y.values.end(), values.begin());

    // Allocate on device memory and copy 
    hipMalloc(&dev_ptr, num_values * b_vecs * sizeof(double));
    copy_kernel<<<tblocks,blocksize>>>(y.dev_ptr, dev_ptr, num_values * b_vecs);
}

/**************************************************************
*****   Print Vector
**************************************************************
***** Prints all nonzero elements in vector
*****
***** Parameters
***** -------------
***** vec_name : const char* (optional)
*****    Name to be printed.  Default prints Vec[%d] = %e.
**************************************************************/
void Vector::print(const char* vec_name)
{
    // Leaving alone -- assuming user will copy from device
    // first then print
    index_t offset;
    printf("Size = %d\n", num_values);
    for (int j = 0; j < b_vecs; j++)
    {
        offset = j * num_values;
        for (int i = 0; i < num_values; i++)
        {
            if (fabs(values[i + offset]) > zero_tol)
                printf("%s[%d] = %e\n", vec_name, j, i, values[i + offset]);
        }
    }
}

/**************************************************************
*****   Vector Element Access
**************************************************************
***** Function overload for element access
*****
***** Returns
***** ------------
***** data_t& element at position passed
**************************************************************/
data_t& Vector::operator[](const int index)
{
    // Leaving alone -- assuming user will copy from device
    // first then print
    return values[index];
}

/**************************************************************
*****   Vector Copy from Device 
**************************************************************
***** Function that copies vector values from device into
***** host vector array
**************************************************************/
void Vector::copy_from_device()
{
    hipMemcpy(dev_ptr, values.data(), values.size()*sizeof(double), hipMemcpyHostToDevice);
}

/**************************************************************
*****   Vector Copy from Host 
**************************************************************
***** Function that copies vector values from host into
***** device vector array
**************************************************************/
void Vector::copy_from_host()
{
    hipMemcpy(values.data(), dev_ptr, values.size()*sizeof(double), hipMemcpyDeviceToHost);
}

/**************************************************************
*****   Vector Append
**************************************************************
***** Appends P to the Vector by adding P as additional 
***** vectors in the Vector and increases the block size 
*****
***** Parameters 
***** ------------
***** P : Vector&
*****    The Vector to append 
**************************************************************/
void Vector::append(Vector& P)
{
    // RESIZE ON DEVICE VECTOR
    // STORE NEW VALUES IN APPROPRIATE PLACE ON DEVICE
    values.resize(num_values * (b_vecs + P.b_vecs));
    std::copy(P.values.begin(), P.values.end(), values.begin() + (num_values * b_vecs));
    b_vecs += P.b_vecs;
}

/**************************************************************
*****   Vector Split 
**************************************************************
***** Splits the vector into t b_vecs
*****
***** Parameters 
***** ------------
***** W : Vector&
*****    The Vector to contain the resulting split Vector
***** t : int
*****    The number of b_vecs to split the Vector into
***** i : int
*****    The index of the Vector in W that should contain the
*****    the calling Vector's values.
**************************************************************/
void Vector::split(Vector& W, int t, int i)
{
    // COPY VECTOR FROM DEV 
    // SPLIT VECTOR ON HOST	
    W.b_vecs = t;
    W.resize(num_values);
    W.set_const_value(0.0);
    std::copy(values.begin(), values.end(), W.values.begin() + (num_values * i));
    // RESIZE VECTOR ON DEV
    // COPY VECTOR BACK TO DEV
}

/**************************************************************
*****   Vector Split Range 
**************************************************************
***** Splits the vector into t b_vecs
***** Splitting the values in the vector across the vectors
***** from block index start to block index stop 
*****
***** Parameters 
***** ------------
***** W : Vector&
*****    The Vector to contain the resulting split Vector
***** t : int
*****    The number of b_vecs to split the Vector into
***** start : int
*****    The index of the Vector in W that should contain the
*****    first portion of the calling Vector's values.
**************************************************************/
void Vector::split_range(Vector& W, int t, int start)
{
    // COPY VECTOR FROM DEV 
    // SPLIT VECTOR ON HOST	
    W.b_vecs = t;
    W.resize(num_values);
    W.set_const_value(0.0);

    for (int i = 0; i < num_values; i++)
    {
        W.values[start*num_values + i] = values[i];
        start = (start + 1) % t;
    }
    // RESIZE VECTOR ON DEV
    // COPY VECTOR BACK TO DEV
}

/**************************************************************
*****   Vector Split Contiguous 
**************************************************************
***** Splits the vector into t b_vecs
***** Splitting the values in the vector across the vectors
***** in equal sized contiguous chunks 
*****
***** Parameters 
***** ------------
***** W : Vector&
*****    The Vector to contain the resulting split Vector
***** t : int
*****    The number of b_vecs to split the Vector into
***** first_global_index : int
*****    The corresponding global index of the first index
*****    in this vector 
**************************************************************/
void Vector::split_contig(Vector& W, int t, int first_global_index, int glob_vals)
{
    // COPY VECTOR FROM DEV 
    // SPLIT VECTOR ON HOST	
    int glob_index, bvec, pos_in_bvec, end;
    int chunk_size = glob_vals / t;

    W.b_vecs = t;
    W.resize(num_values);
    W.set_const_value(0.0);

    for (int i = 0; i < num_values; i+= chunk_size)
    {
        glob_index = i + first_global_index;
        bvec = glob_index / t;
        if (i + chunk_size > num_values) end = num_values;
        else end = chunk_size;
        for (int j = 0; j < chunk_size; j++)
        {
            W.values[bvec*num_values + bvec*chunk_size + j] = values[i + j];
        }
    }
    
    // RESIZE VECTOR ON DEV
    // COPY VECTOR BACK TO DEV
}

